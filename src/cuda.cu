#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#define TPB 21//threads per block
#define BLOCKS 1

typedef struct{
  uint8_t r, g, b;
}pixel_t;


__device__ void computeEnergyShared(pixel_t * img, int64_t  w, int64_t h){
  extern __shared__  int32_t energy[];

  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  int idxPerThread = max(1, (int)(w*h/nThreads));
  int underCompute = (w*h) - (idxPerThread*nThreads);


  int begin = max((int)(idxPerThread * indo), 0);
  begin += indo < underCompute ? indo: underCompute;
  idxPerThread += !!(indo < underCompute);
  
  int end = min((int)(begin + idxPerThread), (int)(w*h));

  //printf("%6i %6i %6i %6i %6i %6i\n", indo, nThreads, idxPerThread, begin, end);
  
  for(int i = begin; i < end; i++){
    energy[i] =
      (img[i-1].r - img[i+1].r) * (img[i-1].r - img[i+1].r) +
      (img[i-1].g - img[i+1].g) * (img[i-1].g - img[i+1].g) +
      (img[i-1].b - img[i+1].b) * (img[i-1].b - img[i+1].b);
  }
  
  /* for(int y = 0; y < h; y++){ */
  /*   for(int x = 0; x < w; x++) */
  /*     printf("%8i", energy[(x+y*w)]); */
  /*   printf("\n"); */
  /* } */

  
}
__global__ void computeEnergy(pixel_t * img, int32_t * energy, int64_t  w, int64_t h){
  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  int idxPerThread = max(1, (int)(w*h/nThreads));
  int underCompute = max((int)((w*h) - (idxPerThread*nThreads)), 0);


  int begin = max((int)(idxPerThread * indo), 0);
  begin += indo < underCompute ? indo: underCompute;
  idxPerThread += !!(indo < underCompute);
  
  int end = min((int)(begin + idxPerThread), (int)(w*h));

  /* printf("%6i %6i %6i %6i %6i %6i\n", */
  /* 	 indo, nThreads, idxPerThread, begin, end, indo<underCompute); */
  
  for(int i = begin; i < end; i++){
    energy[i] =
      (img[i-1].r - img[i+1].r) * (img[i-1].r - img[i+1].r) +
      (img[i-1].g - img[i+1].g) * (img[i-1].g - img[i+1].g) +
      (img[i-1].b - img[i+1].b) * (img[i-1].b - img[i+1].b);
  }
}

__global__ void computeEnergySum(int32_t * energy, int32_t * energySum, int64_t  w, int64_t h){
  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  int idxPerThread = max(1, (int)(w/nThreads));
  int underCompute = max((int)((w) - (idxPerThread*nThreads)), 0);


  int begin = max((int)(idxPerThread * indo), 0);
  begin += indo < underCompute ? indo: underCompute;
  idxPerThread += !!(indo < underCompute);
  
  int end = min((int)(begin + idxPerThread), (int)(w));
  
  printf("%6i %6i %6i %6i %6i %6i\n",
  	 indo, nThreads, idxPerThread, begin, end, indo<underCompute);

  int32_t * res = energySum;
  
  for(int y=1; y < h; ++y){
    if(begin < end){
      int x   = w * y;
      int xu0 = w * (y-1);
      
      int i = 0;
      if(end == w){
	res[x+w-1] += min(res[xu0+w-1-1], res[xu0+w-1]);
	i++;
      }
      if(begin == 0){
	res[x] += min(res[xu0], res[xu0+1]);
	i++; x++; xu0++;
      }
      
      x   += begin;
      xu0 += begin; 
      
      for(x, xu0; i < (end-begin); ++x, ++xu0, i++)
	res[x] += min(min(res[xu0-1], res[xu0]), res[xu0+1]);
    } 
    __syncthreads();
  }
}
__global__ void seamCarving(pixel_t * img, int32_t * energy, int32_t * energySum, int64_t  w, int64_t h){
  /* computeEnergy(img, energy, w, h); */
  /* computeEnergySum(energy, energySum, w, h); */
}


extern "C" void cudaProxy(uint8_t* h_img, int64_t w, int64_t h){
  /*
    memory start
  */  
  pixel_t *d_img;
  int32_t *d_energy;
  int32_t *d_energySum;
  int32_t h_energy[w*h];
  int32_t h_energySum[w*h];
  
  checkCudaErrors( hipMalloc((void **)&d_img,       sizeof(pixel_t)*w*h+1 ) );
  checkCudaErrors( hipMalloc((void **)&d_energy,    sizeof(int32_t)*w*h ) );
  checkCudaErrors( hipMalloc((void **)&d_energySum, sizeof(int32_t)*w*h ) );
  checkCudaErrors( hipMemcpy(d_img, h_img, sizeof(pixel_t)*w*h, hipMemcpyHostToDevice) );
  
  /*
    memory stop
  */
  
  
  float elapsed=0;
  hipEvent_t start, stop;

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors( hipEventRecord(start, 0));
  /*
    Run kernel
  */
  //strumienie moga przyspieszyc
  //child kerlen - lepsza organizacja
  //czy uzywajac shared mozna przyspieczyc 
  
  //int shared_mem_size = w*h *sizeof(int32_t);
  //seamCarving<<<BLOCKS, TPB, shared_mem_size>>>(d_img, w, h);
  //seamCarving<<<BLOCKS, TPB>>>(d_img, d_energy, d_energySum, w, h);
  computeEnergy<<<BLOCKS, TPB>>>(d_img, d_energy, w, h);
  checkCudaErrors( hipMemcpy(d_energySum, d_energy,
			      sizeof(int32_t)*w*h, hipMemcpyDeviceToDevice) );
  computeEnergySum<<<BLOCKS, 1>>>(d_energy, d_energySum, w, h);
  hipDeviceSynchronize();
  /*
    Stop kernel 
  */
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize (stop) );

  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop) );

  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  printf("The elapsed time in gpu was %.2f ms\n", elapsed);

  /*
    memory free start
  */
  
  checkCudaErrors( hipMemcpy(&h_energy, d_energy, sizeof(int32_t)*w*h, hipMemcpyDeviceToHost) );
  checkCudaErrors( hipMemcpy(&h_energySum, d_energySum, sizeof(int32_t)*w*h, hipMemcpyDeviceToHost) );
  checkCudaErrors( hipFree(d_img) );

  for(int y = 0; y < h; y++){
    for(int x = 0; x < w; x++)
      printf("%8i", h_energySum[(x+y*w)]);
    printf("\n");
  }

  
  /*
    memory free stop
   */
  
  hipDeviceReset();
}

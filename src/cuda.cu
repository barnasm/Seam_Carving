#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#define TPB 1 //threads per block
#define BLOCKS 1

typedef struct{
  uint8_t r, g, b;
}pixel_t;


__global__ void seamCarving(pixel_t * img, int64_t  w, int64_t h){
  extern __shared__  int32_t energy[];

  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;
  int idxPreThread = w*h/nThreads;
  int begin = max((int)(idxPreThread * indo), 0);
  int end = min((int)(begin + idxPreThread), (int)(w*h));
  
  for(int i = begin; i < end; i++){
    energy[i] =
      (img[i-1].r - img[i+1].r) * (img[i-1].r - img[i+1].r) +
      (img[i-1].g - img[i+1].g) * (img[i-1].g - img[i+1].g) +
      (img[i-1].b - img[i+1].b) * (img[i-1].b - img[i+1].b);
  }
  
  printf("%i %i %i\n", (int)img[0].r, (int)img[0].g, (int)img[0].b);
  printf("size of img=%i  w=%li h=%li \n", sizeof(*img), w, h);
  //printf("size of img=%i  w=%lu h=%lu \n", sizeof(*img), width, height);

  //int32_t energy[w*h];

  for(int y = 0; y < h; y++){
    for(int x = 0; x < w; x++)
      printf("%4i%4i%4i     ", (int)img[(x+y*w)].r , (int)img[(x+y*w)].g, (int)img[(x+y*w)].b);
    //std::cout << std::setw(8) << energyTable(x,y);
    printf("\n");
  }

  for(int y = 0; y < h; y++){
    for(int x = 0; x < w; x++)
      printf("%8i", energy[(x+y*w)]);
    printf("\n");
  }

  
}

void cudaProxy(uint8_t* h_img, int64_t w, int64_t h){
  /*
    memory start
  */  
  pixel_t *d_img;
  
  checkCudaErrors( hipMalloc((void **)&d_img, sizeof(pixel_t)*w*h ) );
  checkCudaErrors( hipMemcpy(d_img, h_img, sizeof(pixel_t)*w*h, hipMemcpyHostToDevice) );
  
  for(int y = 0; y < h; y++){
    for(int x = 0; x < w; x++)
      printf("%4i%4i%4i     ", (int)h_img[3*(x+y*w)] , (int)h_img[3*(x+y*w)+1], (int)h_img[3*(x+y*w)+2]);
    //std::cout << std::setw(8) << energyTable(x,y);
    printf("\n");
  }


  /*
    memory stop
  */
  
  
  float elapsed=0;
  hipEvent_t start, stop;

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors( hipEventRecord(start, 0));
  /*
    Run kernel
  */
  int shared_mem_size = w*h *sizeof(int32_t);
  seamCarving<<<BLOCKS, TPB, shared_mem_size>>>(d_img, w, h);
  hipDeviceSynchronize();
  /*
    Stop kernel 
  */
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize (stop) );

  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop) );

  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  printf("The elapsed time in gpu was %.2f ms\n", elapsed);

  /*
    memory free start
  */
  
  //checkCudaErrors( hipMemcpy(h_img, d_img, sizeof(h_img), hipMemcpyDeviceToHost) );
  checkCudaErrors( hipFree(d_img) );
  
  /*
    memory free stop
   */
  
  hipDeviceReset();
}

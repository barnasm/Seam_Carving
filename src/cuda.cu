#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#define TPB 512//threads per block
#define BLOCKS 4

typedef struct{
  uint8_t r, g, b;
}pixel_t;


__device__ void computeEnergyShared(pixel_t * img, int64_t  w, int64_t h){
  extern __shared__  int32_t energy[];

  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  int idxPerThread = max(1, (int)(w*h/nThreads));
  int underCompute = (w*h) - (idxPerThread*nThreads);


  int begin = max((int)(idxPerThread * indo), 0);
  begin += indo < underCompute ? indo: underCompute;
  idxPerThread += !!(indo < underCompute);
  
  int end = min((int)(begin + idxPerThread), (int)(w*h));

  //printf("%6i %6i %6i %6i %6i %6i\n", indo, nThreads, idxPerThread, begin, end);
  
  for(int i = begin; i < end; i++){
    energy[i] =
      (img[i-1].r - img[i+1].r) * (img[i-1].r - img[i+1].r) +
      (img[i-1].g - img[i+1].g) * (img[i-1].g - img[i+1].g) +
      (img[i-1].b - img[i+1].b) * (img[i-1].b - img[i+1].b);
  }
  
  /* for(int y = 0; y < h; y++){ */
  /*   for(int x = 0; x < w; x++) */
  /*     printf("%8i", energy[(x+y*w)]); */
  /*   printf("\n"); */
  /* } */

  
}
__global__ void computeEnergy(pixel_t * img, int32_t * energy, int64_t  w, int64_t h){
  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  int idxPerThread = max(1, (int)(w*h/nThreads));
  int underCompute = max((int)((w*h) - (idxPerThread*nThreads)), 0);


  int begin = max((int)(idxPerThread * indo), 0);
  begin += indo < underCompute ? indo: underCompute;
  idxPerThread += !!(indo < underCompute);
  
  int end = min((int)(begin + idxPerThread), (int)(w*h));

  /* printf("%6i %6i %6i %6i %6i %6i\n", */
  /* 	 indo, nThreads, idxPerThread, begin, end, indo<underCompute); */
  
  for(int i = begin; i < end; i++){
    energy[i] =
      (img[i-1].r - img[i+1].r) * (img[i-1].r - img[i+1].r) +
      (img[i-1].g - img[i+1].g) * (img[i-1].g - img[i+1].g) +
      (img[i-1].b - img[i+1].b) * (img[i-1].b - img[i+1].b);
  }
}

__global__ void computeEnergySum(int32_t * energy, int32_t * energySum, int64_t  w, int64_t h){
  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  int idxPerThread = max(1, (int)(w/nThreads));
  int underCompute = max((int)((w) - (idxPerThread*nThreads)), 0);


  int begin = max((int)(idxPerThread * indo), 0);
  begin += indo < underCompute ? indo: underCompute;
  idxPerThread += !!(indo < underCompute);
  
  int end = min((int)(begin + idxPerThread), (int)(w));
  
  /* printf("%6i %6i %6i %6i %6i %6i\n", */
  /* 	 indo, nThreads, idxPerThread, begin, end, indo<underCompute); */

  int32_t * res = energySum;
  
  for(int y=1; y < h; ++y){
    if(begin < end){
      int x   = w * y;
      int xu0 = w * (y-1);
      
      int i = 0;
      if(end == w){
	res[x+w-1] += min(res[xu0+w-1-1], res[xu0+w-1]);
	i++;
      }
      if(begin == 0){
	res[x] += min(res[xu0], res[xu0+1]);
	i++; x++; xu0++;
      }
      
      
      for(x += begin, xu0 += begin; i < (end-begin); ++x, ++xu0, i++)
	res[x] += min(min(res[xu0-1], res[xu0]), res[xu0+1]);
    } 
    __syncthreads();
  }
}

__device__ int find_min(int32_t * arr, int n){
    int minIdx = 0;
    for(int i = 0; i < n; i++)
      if (arr[i] < arr[minIdx])
	minIdx = i;
    return minIdx;
  };


__global__ void findMinPath(int32_t * energy, int32_t * energySum, int32_t * removedPixels, int64_t  w, int64_t h){

  
  int32_t * res = energySum;
    
  int y = w * (h-1);
  int x = find_min(&res[y], w);
    
  for(int i=1; i < h; i++){
    int off = x;
    //removedPixels[off + (w*(h-i))] = true;
    removedPixels[h-i] = off;
    y -= w;
    
     if(off == 0)
       x = find_min(&res[y], 2);
     else if(off == w-1)
       x += find_min(&res[y+off-1], 2) -1;
     else
       x += find_min(&res[y+off-1], 3) -1;
  }
  int off = x;
  //removedPixels[off] = true;
  removedPixels[0] = off;
}


__global__ void removeSeam(int32_t * removedPixels, pixel_t * img, pixel_t * img_res, int64_t  w, int64_t h){
  int indo = threadIdx.x + blockIdx.x*blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  int idxPerThread = max(1, (int)(w*h/nThreads));
  int underCompute = max((int)((w*h) - (idxPerThread*nThreads)), 0);


  int begin = max((int)(idxPerThread * indo), 0);
  begin += indo < underCompute ? indo: underCompute;
  idxPerThread += !!(indo < underCompute);
  
  int end = min((int)(begin + idxPerThread), (int)(w*h));

  /* printf("%6i %6i %6i %6i %6i %6i\n", */
  /* 	 indo, nThreads, idxPerThread, begin, end, indo<underCompute); */

  
  int64_t i = begin;
  for (int y=i/(w-1); y<h; ++y)
    for (int x= i==begin ? i%(w-1) : 0; x<w-1; ++x, i++){
      if(i >= end) return;
      img_res[x+(w-1)*y] = img[x + w*y + ((x >= removedPixels[y])? 1:0)];
    }
}


__global__ void seamCarving(pixel_t * img, int32_t * energy, int32_t * energySum, int64_t  w, int64_t h){
  /* computeEnergy(img, energy, w, h); */
  /* computeEnergySum(energy, energySum, w, h); */
}


extern "C" void cudaProxy(uint8_t* h_img, uint8_t* h_img_res, int64_t w, int64_t h, int64_t N){
  /*
    memory start
  */  
  pixel_t *d_img;
  pixel_t *d_img_res;
  int32_t *d_energy;
  int32_t *d_energySum;
  int32_t *d_removedPixels;
  //int32_t h_energy[w*h];
  //int32_t h_energySum[w*h];
  //int32_t  h_removedPixels[h];

  checkCudaErrors( hipMalloc((void **)&d_img,         sizeof(pixel_t)*w*h+2*sizeof(pixel_t) ) );
  checkCudaErrors( hipMalloc((void **)&d_energy,        sizeof(int32_t)*w*h ) );
  checkCudaErrors( hipMalloc((void **)&d_energySum,     sizeof(int32_t)*w*h ) );
  checkCudaErrors( hipMalloc((void **)&d_removedPixels, sizeof(int32_t)*h ) );


  checkCudaErrors( hipMemcpy(&d_img[1], h_img, sizeof(pixel_t)*w*h, hipMemcpyHostToDevice) );
  /*
    memory stop
  */
  
  
  float elapsed=0;
  hipEvent_t start, stop;

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors( hipEventRecord(start, 0));
  /*
    Run kernel
  */
  //strumienie moga przyspieszyc
  //child kerlen - lepsza organizacja
  //czy uzywajac shared mozna przyspieczyc 
  
  //int shared_mem_size = w*h *sizeof(int32_t);
  //seamCarving<<<BLOCKS, TPB, shared_mem_size>>>(d_img, w, h);
  //seamCarving<<<BLOCKS, TPB>>>(d_img, d_energy, d_energySum, w, h);

  
  for(int i = 0; i < N; i++){

    checkCudaErrors( hipMalloc((void **)&d_img_res, sizeof(pixel_t)*(w-1)*h+2*sizeof(pixel_t)) );
    
    pixel_t *d_img_off     = &d_img[1];
    pixel_t *d_img_res_off = &d_img_res[1];
    
    computeEnergy<<<BLOCKS, TPB>>>(d_img_off, d_energy, w, h);
    checkCudaErrors( hipMemcpy(d_energySum, d_energy, sizeof(int32_t)*w*h, hipMemcpyDeviceToDevice) );
    computeEnergySum<<<1, TPB>>>(d_energy, d_energySum, w, h);
    findMinPath<<<1, 1>>>(d_energy, d_energySum, d_removedPixels, w, h);
    removeSeam<<<BLOCKS, TPB>>>(d_removedPixels, d_img_off, d_img_res_off, w, h);

    checkCudaErrors( hipFree(d_img) );

    d_img = d_img_res;
    w--;

  }
  hipDeviceSynchronize();
  /*
    Stop kernel 
  */
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize (stop) );

  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop) );

  
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  printf("The elapsed time in gpu was %.2f ms\n", elapsed);

  /*
    memory free start
  */

  /* for(int i = 0; i < h; i++) */
  /*   checkCudaErrors( hipMemcpy(&h_img_res[i*w], &d_img[(w+N)*i], sizeof(pixel_t)*w, hipMemcpyDeviceToHost) ); */

  checkCudaErrors( hipMemcpy(h_img_res, &d_img[1], sizeof(pixel_t)*w*h, hipMemcpyDeviceToHost) ); 

  
  //checkCudaErrors( hipMemcpy(&h_energy, d_energy, sizeof(int32_t)*w*h, hipMemcpyDeviceToHost) );
  //checkCudaErrors( hipMemcpy(&h_energySum, d_energySum, sizeof(int32_t)*w*h, hipMemcpyDeviceToHost) );
  //checkCudaErrors( hipMemcpy(&h_removedPixels, d_removedPixels, sizeof(int8_t)*w*h, hipMemcpyDeviceToHost) );
  checkCudaErrors( hipFree(d_img) );
  checkCudaErrors( hipFree(d_energy) );
  checkCudaErrors( hipFree(d_energySum) );
  checkCudaErrors( hipFree(d_removedPixels) );

  
  /* for(int y = 0; y < h; y++){ */
  /*   for(int x = 0; x < w; x++) */
  /*     printf("%8i", (int)h_removedPixels[(x+y*w)]); */
  /*   printf("\n"); */
  /* } */

  
  /*
    memory free stop
   */
  
  hipDeviceReset();
}
